#include "hip/hip_runtime.h"
/*
Fractal code for CS 4380 / CS 5351

Copyright (c) 2020 Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source or binary form, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdio>
#include <cmath>
#include <algorithm>
#include <sys/time.h>
#include "BMP43805351.h"
#include <hip/hip_runtime.h>



static const float ThreadsPerBlock = 1024;

static __global__ void fractalKernel(const int width, const int frames, unsigned char* const pic)
{
   
   


  const float Delta = 0.00304f;
  const float xMid = -0.055846456f;
  const float yMid = -0.668311119f;

  // compute pixels of each frame
 // double delta = Delta;
//  for (int frame = 0; frame < frames; frame++) {  // frames

   const int pixels = frames * width * width;
   const int i = threadIdx.x + blockIdx.x * blockDim.x;
   if(i < pixels){
      const int frame = i / (width * width);
      const int row = (i / width) % width;
      const int col = i % width;
 
      const float delta = Delta * powf(0.975f, frame);   

    const float xMin = xMid - delta;
    const float yMin = yMid - delta;
    const float dw = 2.0f * delta / width;
//    for (int row = 0; row < width; row++) {  // rows
      const float cy = yMin + row * dw;
//      for (int col = 0; col < width; col++) {  // columns
        const float cx = xMin + col * dw;
        float x = cx;
        float y = cy;
        float x2, y2;
        int count = 256;
        do {
          x2 = x * x;
          y2 = y * y;
          y = 2.0f * x * y + cy;
          x = x2 - y2 + cx;
          count--;
        } while ((count > 0) && ((x2 + y2) <= 5.0f));
        pic[frame * width * width + row * width + col] = (unsigned char)count;
     // }
   // }
 //   delta *= 0.975;
  }
}


  static void CheckCuda(){
     hipError_t e;
     hipDeviceSynchronize();
     if(hipSuccess != (e = hipGetLastError())){
        fprintf(stderr, "CUDA error %d: %s\n", e, hipGetErrorString(e));
        exit(-1);
     }
  }



int main(int argc, char *argv[])
{
  printf("Fractal v2.1\n");

  // check command line
  if (argc != 3) {fprintf(stderr, "USAGE: %s frame_width number_of_frames\n", argv[0]); exit(-1);}
  const int width = atoi(argv[1]);
  if (width < 8) {fprintf(stderr, "ERROR: frame_width must be at least 8\n"); exit(-1);}
  const int frames = atoi(argv[2]);
  if (frames < 1) {fprintf(stderr, "ERROR: number_of_frames must be at least 1\n"); exit(-1);}
  printf("frames: %d\n", frames);
  printf("width: %d\n", width);

  // allocate picture array
 
   const int pixels = frames  * width * width;
   unsigned char * dev_pic;
   const int size = pixels * sizeof(unsigned char);
   hipMalloc((void**)&dev_pic, size);

   unsigned char* pic = new unsigned char [frames * width * width]; // same as pixels

   if(hipSuccess != hipMemcpy(dev_pic, pic, size, hipMemcpyHostToDevice)) {fprintf(stderr, "copying to device failed\n"); exit(-1);}

  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // execute timed code
//  fractal(width, frames, pic);

   fractalKernel<<<(pixels + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(width, frames, dev_pic);
   hipDeviceSynchronize();


  // end time
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0;
  printf("compute time: %.6f s\n", runtime);

  CheckCuda();

  if(hipSuccess != hipMemcpy(pic, dev_pic, size, hipMemcpyDeviceToHost)) {fprintf(stderr, "copying from device failed\n"); exit(-1);}


  

  // write result to BMP files
  if ((width <= 256) && (frames <= 64)) {
    for (int frame = 0; frame < frames; frame++) {
      BMP24 bmp(0, 0, width, width);
      for (int y = 0; y < width; y++) {
        for (int x = 0; x < width; x++) {
          bmp.dot(x, y, pic[frame * width * width + y * width + x] * 0x010101);
        }
      }
      char name[32];
      sprintf(name, "fractal%d.bmp", frame + 1000);
      bmp.save(name);
    }
  }

  // clean up
  delete [] pic;
  hipFree(dev_pic);
  return 0;
}

