/*
Collatz code for CS 4380 / CS 5351

Copyright (c) 2020 Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source or binary form, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher

//Partner: Luis Tovar

*/

#include <cstdio>
#include <algorithm>
#include <sys/time.h>

#include <hip/hip_runtime.h>

static const int ThreadsPerBlock = 1024;

//static int collatz(const long bound)

static __global__ void collatzKernel(const long range, int* maxlen)
{
   const long i = threadIdx.x + blockIdx.x * (long)blockDim.x;
   long val = i + 1;
   int len = 1;

  // compute sequence lengths
//  int maxlen = 0;
//  for (long i = 1; i <= bound; i++) {
//    long val = i;
//    int len = 1;

   if(i < range){
    while (val != 1) {
      len++;
      if ((val % 2) == 0) {
        val /= 2;  // even
      } else {
        val = 3 * val + 1;  // odd
      }
    }
    if(*maxlen < len)
       atomicMax(maxlen,len);
  }
}


   static void CheckCuda(){
      hipError_t e;
      hipDeviceSynchronize();
      if( hipSuccess != (e = hipGetLastError())){
         fprintf(stderr, "CUDA error %d: %s\n", e, hipGetErrorString(e));
         exit(-1);
      }
    }   



int main(int argc, char *argv[])
{
  printf("Collatz v1.4\n");

  // check command line
  if (argc != 2) {fprintf(stderr, "USAGE: %s upper_bound\n", argv[0]); exit(-1);}
  const long bound = atol(argv[1]);
  if (bound < 1) {fprintf(stderr, "ERROR: upper_bound must be at least 1\n"); exit(-1);}
  printf("upper bound: %ld\n", bound);


   int* dev_maxlen;
   const int size = sizeof(int);
   hipMalloc((void**)&dev_maxlen, size);


   int* host_maxlen = new int;
   *host_maxlen = 0;

   if( hipSuccess != hipMemcpy(dev_maxlen, host_maxlen, size, hipMemcpyHostToDevice)){fprintf(stderr, "copying to device failed\n"); exit(-1);}
  
  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // execute timed code
 // const int maxlen = collatz(bound);

   collatzKernel<<<(ThreadsPerBlock + bound - 1)/ThreadsPerBlock, ThreadsPerBlock>>>(bound, dev_maxlen);
   hipDeviceSynchronize();

  // end time
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0;
  printf("compute time: %.6f s\n", runtime);
  CheckCuda();

  if(hipSuccess != hipMemcpy(host_maxlen, dev_maxlen, size, hipMemcpyDeviceToHost)){fprintf(stderr, "copying from device failed\n"); exit(-1);}


  // print result
  printf("longest sequence length: %d elements\n", host_maxlen);
 
  delete host_maxlen;
  hipFree(dev_maxlen);
  return 0;
}
