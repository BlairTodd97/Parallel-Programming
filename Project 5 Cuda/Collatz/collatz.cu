#include "hip/hip_runtime.h"
/*
Collatz code for CS 4380 / CS 5351

Copyright (c) 2020 Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source or binary form, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher

//add names here

*/

#include <cstdio>
#include <algorithm>
#include <sys/time.h>

#include <hip/hip_runtime.h>

static const int ThreadsPerBlock = 512;

//static int collatz(const long bound)

static __global__ void collatzKernal(const long range, int* maxlen)
{
   const long i = threadIdx.x + blockIdx.x * (long)blockDim.x;
   long val = i + 1;
   int len = 1;

  // compute sequence lengths
//  int maxlen = 0;
//  for (long i = 1; i <= bound; i++) {
//    long val = i;
//    int len = 1;

   if(i < range){
    while (val != 1) {
      len++;
      if ((val % 2) == 0) {
        val /= 2;  // even
      } else {
        val = 3 * val + 1;  // odd
      }
    }
    if(*maxlen < len)
       atomicMax(maxlen,len);
  }
}

int main(int argc, char *argv[])
{
  printf("Collatz v1.4\n");

  // check command line
  if (argc != 2) {fprintf(stderr, "USAGE: %s upper_bound\n", argv[0]); exit(-1);}
  const long bound = atol(argv[1]);
  if (bound < 1) {fprintf(stderr, "ERROR: upper_bound must be at least 1\n"); exit(-1);}
  printf("upper bound: %ld\n", bound);


   int* dev_maxlen;
   const int size = sizeof(int);
   hipMalloc((void**)&dev_maxlen,size);


   int* host_maxlen = new int;
   *h_maxlen = 0;

   hipMemcpy(dev_maxlen, host_maxlen, size, cudaMemcpyHosttoDevice);
  
  // start time
  timeval start, end;
  gettimeofday(&start, NULL);

  // execute timed code
 // const int maxlen = collatz(bound);

   collatzKernel<<<(ThreadsPerBlock + bound - 1)/ThreadsPerBlock, ThreadsPerBlock>>>(bound, d_maxlen);
   hipDeviceSynchronize();

  // end time
  gettimeofday(&end, NULL);
  const double runtime = end.tv_sec - start.tv_sec + (end.tv_usec - start.tv_usec) / 1000000.0;
  printf("compute time: %.6f s\n", runtime);

  // print result
  printf("longest sequence length: %d elements\n", maxlen);
 
  delete h_maxlen;
  hipFree(dev_maxlen);
  return 0;
}
